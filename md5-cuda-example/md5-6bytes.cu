#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "book.h"
// #include "libs/cuPrintf.cu"
#include "md5.h"


#define T_MASK ((md5_word_t)~0)
#define T1    0xd76aa478
#define T2    0xe8c7b756
#define T3    0x242070db
#define T4    0xc1bdceee
#define T5    0xf57c0faf
#define T6    0x4787c62a
#define T7    0xa8304613
#define T8    0xfd469501
#define T9    0x698098d8
#define T10   0x8b44f7af
#define T11   0xffff5bb1
#define T12   0x895cd7be
#define T13   0x6b901122
#define T14   0xfd987193
#define T15   0xa679438e
#define T16   0x49b40821
#define T17   0xf61e2562
#define T18   0xc040b340
#define T19   0x265e5a51
#define T20   0xe9b6c7aa
#define T21   0xd62f105d
#define T22   0x02441453
#define T23   0xd8a1e681
#define T24   0xe7d3fbc8
#define T25   0x21e1cde6
#define T26   0xc33707d6
#define T27   0xf4d50d87
#define T28   0x455a14ed
#define T29   0xa9e3e905
#define T30   0xfcefa3f8
#define T31   0x676f02d9
#define T32   0x8d2a4c8a
#define T33   0xfffa3942
#define T34   0x8771f681
#define T35   0x6d9d6122
#define T36   0xfde5380c
#define T37   0xa4beea44
#define T38   0x4bdecfa9
#define T39   0xf6bb4b60
#define T40   0xbebfbc70
#define T41   0x289b7ec6
#define T42   0xeaa127fa
#define T43   0xd4ef3085
#define T44   0x04881d05
#define T45   0xd9d4d039
#define T46   0xe6db99e5
#define T47   0x1fa27cf8
#define T48   0xc4ac5665
#define T49   0xf4292244
#define T50   0x432aff97
#define T51   0xab9423a7
#define T52   0xfc93a039
#define T53   0x655b59c3
#define T54   0x8f0ccc92
#define T55   0xffeff47d
#define T56   0x85845dd1
#define T57   0x6fa87e4f
#define T58   0xfe2ce6e0
#define T59   0xa3014314
#define T60   0x4e0811a1
#define T61   0xf7537e82
#define T62   0xbd3af235
#define T63   0x2ad7d2bb
#define T64   0xeb86d391


#define STR_SIZE 6
// If the string has 4 chars, it has a size of 32 bits
// X is a unsigned int pointer, it points to 32 bit chuncks of data
// X[0] will be the whole 4 byte string
// the byte right after the string has the value 0x80, which was translating to 128 since the first nibble is 0 and the second is 8
// a byte with the second nibble as 8 is the decimal 128 --> 1000 0000
// the formula for X_14 is STR_SIZE << 3 as long the string has less than 32 bytes
// after that the formula is (STR_SIZE - 32) << 3 and X_15 will be 1
// We'll be focusing on string less than 32 bytes for now
#define X_1 128
// #define X_14 (STR_SIZE << 3)
#define X_14 (STR_SIZE << 3)
#define ZERO 0

#define WORD_SIZE (STR_SIZE)

/* Round 1. */
/* Let [abcd k s i] denote the operation
   a = b + ((a + F(b,c,d) + X[k] + T[i]) <<< s).
*/
/* Do the following 16 operations. */
// For the first operation the values of a,b,c,d are static
// ~b == d
// Still need to run some calculations and read the md5 RFC to understand what's going on
// For now all we know is that we can replace the first set of calculations by d
// t = a + ((b & c) | (~b & d)) + x_0 +  T1;   a = ((t << 7)   | (t >> (25))) + b; \

#define ROUND_1 \
  t = a + (c) + x_0                  +  T1;   a = ((t << 7)   | (t >> (25))) + b; \
  t = d + ((a & b) | (~a & c)) + x_1 +  T2;   d = ((t << 12)  | (t >> (20))) + a; \
  t = c + ((d & a) | (~d & b)) +        T3;   c = ((t << 17)  | (t >> (15))) + d; \
  t = b + ((c & d) | (~c & a)) +        T4;   b = ((t << 22)  | (t >> (10))) + c; \
  t = a + ((b & c) | (~b & d)) +        T5;   a = ((t << 7)   | (t >> (25))) + b; \
  t = d + ((a & b) | (~a & c)) +        T6;   d = ((t << 12)  | (t >> (20))) + a; \
  t = c + ((d & a) | (~d & b)) +        T7;   c = ((t << 17)  | (t >> (15))) + d; \
  t = b + ((c & d) | (~c & a)) +        T8;   b = ((t << 22)  | (t >> (10))) + c; \
  t = a + ((b & c) | (~b & d)) +        T9;   a = ((t << 7)   | (t >> (25))) + b; \
  t = d + ((a & b) | (~a & c)) +        T10;  d = ((t << 12)  | (t >> (20))) + a; \
  t = c + ((d & a) | (~d & b)) +        T11;  c = ((t << 17)  | (t >> (15))) + d; \
  t = b + ((c & d) | (~c & a)) +        T12;  b = ((t << 22)  | (t >> (10))) + c; \
  t = a + ((b & c) | (~b & d)) +        T13;  a = ((t << 7)   | (t >> (25))) + b; \
  t = d + ((a & b) | (~a & c)) +        T14;  d = ((t << 12)  | (t >> (20))) + a; \
  t = c + ((d & a) | (~d & b)) + X_14 + T15;  c = ((t << 17)  | (t >> (15))) + d; \
  t = b + ((c & d) | (~c & a)) +        T16;  b = ((t << 22)  | (t >> (10))) + c; \


/* Round 2. */
/* Let [abcd k s i] denote the operation
   a = b + ((a + G(b,c,d) + X[k] + T[i]) <<< s).
*/
/* Do the following 16 operations. */
#define ROUND_2 \
  t = a + ((b & d) | (c & ~d)) + x_1 +  T17; a = ((t << 5)  | (t >> (27))) + b; \
  t = d + ((a & c) | (b & ~c)) +        T18; d = ((t << 9)  | (t >> (23))) + a; \
  t = c + ((d & b) | (a & ~b)) +        T19; c = ((t << 14) | (t >> (18))) + d; \
  t = b + ((c & a) | (d & ~a)) + x_0 +  T20; b = ((t << 20) | (t >> (12))) + c; \
  t = a + ((b & d) | (c & ~d)) +        T21; a = ((t << 5)  | (t >> (27))) + b; \
  t = d + ((a & c) | (b & ~c)) +        T22; d = ((t << 9)  | (t >> (23))) + a; \
  t = c + ((d & b) | (a & ~b)) +        T23; c = ((t << 14) | (t >> (18))) + d; \
  t = b + ((c & a) | (d & ~a)) +        T24; b = ((t << 20) | (t >> (12))) + c; \
  t = a + ((b & d) | (c & ~d)) +        T25; a = ((t << 5)  | (t >> (27))) + b; \
  t = d + ((a & c) | (b & ~c)) + X_14 + T26; d = ((t << 9)  | (t >> (23))) + a; \
  t = c + ((d & b) | (a & ~b)) +        T27; c = ((t << 14) | (t >> (18))) + d; \
  t = b + ((c & a) | (d & ~a)) +        T28; b = ((t << 20) | (t >> (12))) + c; \
  t = a + ((b & d) | (c & ~d)) +        T29; a = ((t << 5)  | (t >> (27))) + b; \
  t = d + ((a & c) | (b & ~c)) +        T30; d = ((t << 9)  | (t >> (23))) + a; \
  t = c + ((d & b) | (a & ~b)) +        T31; c = ((t << 14) | (t >> (18))) + d; \
  t = b + ((c & a) | (d & ~a)) +        T32; b = ((t << 20) | (t >> (12))) + c; \


/* Round 3. */
/* Let [abcd k s t] denote the operation
   a = b + ((a + H(b,c,d) + X[k] + T[i]) <<< s).
*/
/* Do the following 16 operations. */
#define ROUND_3 \
  t = a + (b ^ c ^ d) +         T33; a = ((t << 4)  | (t >> (28))) + b; \
  t = d + (a ^ b ^ c) +         T34; d = ((t << 11) | (t >> (21))) + a; \
  t = c + (d ^ a ^ b) +         T35; c = ((t << 16) | (t >> (16))) + d; \
  t = b + (c ^ d ^ a) + X_14 +  T36; b = ((t << 23) | (t >> (9)))  + c; \
  t = a + (b ^ c ^ d) + x_1 +   T37; a = ((t << 4)  | (t >> (28))) + b; \
  t = d + (a ^ b ^ c) +         T38; d = ((t << 11) | (t >> (21))) + a; \
  t = c + (d ^ a ^ b) +         T39; c = ((t << 16) | (t >> (16))) + d; \
  t = b + (c ^ d ^ a) +         T40; b = ((t << 23) | (t >> (9)))  + c; \
  t = a + (b ^ c ^ d) +         T41; a = ((t << 4)  | (t >> (28))) + b; \
  t = d + (a ^ b ^ c) + x_0 +   T42; d = ((t << 11) | (t >> (21))) + a; \
  t = c + (d ^ a ^ b) +         T43; c = ((t << 16) | (t >> (16))) + d; \
  t = b + (c ^ d ^ a) +         T44; b = ((t << 23) | (t >> (9)))  + c; \
  t = a + (b ^ c ^ d) +         T45; a = ((t << 4)  | (t >> (28))) + b; \
  t = d + (a ^ b ^ c) +         T46; d = ((t << 11) | (t >> (21))) + a; \
  t = c + (d ^ a ^ b) +         T47; c = ((t << 16) | (t >> (16))) + d; \
  t = b + (c ^ d ^ a) +         T48; b = ((t << 23) | (t >> (9)))  + c; \


/* Round 4. */
/* Let [abcd k s t] denote the operation
   a = b + ((a + I(b,c,d) + X[k] + T[i]) <<< s).
*/
/* Do the following 16 operations. */
#define ROUND_4 \
  t = a + (c ^ (b | ~d)) + x_0 +  T49; a = ((t << 6)  | (t >> (26))) + b; \
  t = d + (b ^ (a | ~c)) +        T50; d = ((t << 10) | (t >> (22))) + a; \
  t = c + (a ^ (d | ~b)) + X_14 + T51; c = ((t << 15) | (t >> (17))) + d; \
  t = b + (d ^ (c | ~a)) +        T52; b = ((t << 21) | (t >> (11))) + c; \
  t = a + (c ^ (b | ~d)) +        T53; a = ((t << 6)  | (t >> (26))) + b; \
  t = d + (b ^ (a | ~c)) +        T54; d = ((t << 10) | (t >> (22))) + a; \
  t = c + (a ^ (d | ~b)) +        T55; c = ((t << 15) | (t >> (17))) + d; \
  t = b + (d ^ (c | ~a)) + x_1 +  T56; b = ((t << 21) | (t >> (11))) + c; \
  t = a + (c ^ (b | ~d)) +        T57; a = ((t << 6)  | (t >> (26))) + b; \
  t = d + (b ^ (a | ~c)) +        T58; d = ((t << 10) | (t >> (22))) + a; \
  t = c + (a ^ (d | ~b)) +        T59; c = ((t << 15) | (t >> (17))) + d; \
  t = b + (d ^ (c | ~a)) +        T60; b = ((t << 21) | (t >> (11))) + c; \
  t = a + (c ^ (b | ~d)) +        T61; a = ((t << 6)  | (t >> (26))) + b; \
  t = d + (b ^ (a | ~c)) +        T62; d = ((t << 10) | (t >> (22))) + a; \
  t = c + (a ^ (d | ~b)) +        T63; c = ((t << 15) | (t >> (17))) + d; \
  t = b + (d ^ (c | ~a)) +        T64; b = ((t << 21) | (t >> (11))) + c; \

void create_md5_hash_str (const char* word, char* hash_str);
void break_down_hash (char* hash, char* hash_str);
int hex_to_decimal (char c);

int brute_force (char* original_word);



 __constant__ char constant_ascii_codes[64];
 __constant__ int constant_hash_to_break[4];

__global__ void kernel(int* global_word, char offset_1, char offset_2, char offset_3, char offset_4, char offset_5) {

  unsigned int a, b, c, d;
  unsigned int t;

  // Iterate on the 1st letter -> use threadIdx.x
  int x_0 = constant_ascii_codes[threadIdx.x];
  int x_1;

  char cached_ascii_code_1 = constant_ascii_codes[offset_1+threadIdx.y];
  char cached_ascii_code_2 = constant_ascii_codes[offset_2+blockIdx.x];
  char cached_ascii_code_3 = constant_ascii_codes[offset_3+blockIdx.y];
  char cached_ascii_code_4 = constant_ascii_codes[offset_4+blockIdx.z];
  char cached_ascii_code_5 = constant_ascii_codes[offset_5];

  int cached_hash_to_break_1 = constant_hash_to_break[0];
  int cached_hash_to_break_2 = constant_hash_to_break[1];
  int cached_hash_to_break_3 = constant_hash_to_break[2];
  int cached_hash_to_break_4 = constant_hash_to_break[3];

  // Iterate on the 2th letter -> use threadIdx.y
  // Block has 16 threads on the y dimension
  // That means this code will iterate 16 times on the ascii_code values
  x_0 |= cached_ascii_code_1 << 8;

  // Iterate on the 3th letter -> use blockIdx.x
  // Grid has 16 threads on the x dimension and 4 on the y
  // That means that blockIdx.x will iterate from 0-16 and on the host we launch
  // a kernel 4 times increments the offset_2 by 16 each time
  // This way all possible combinations are tested
  x_0 |= cached_ascii_code_2 << 16;

  // Iterate on the 4th letter -> use blockIdx.y
  x_0 |= cached_ascii_code_3 << 24;

  // Iterate on the 5th letter -> use blockIdx.z
  x_1 = cached_ascii_code_4;

  // Iterate on the 6th letter -> launch kernel 64 times for all 5 letter combinations
  x_1 |= cached_ascii_code_5 << 8;

  // Add padding bit
  x_1 |= 0x80 << 16;

  a = 0x67452301;
  b = 0xefcdab89;
  c = 0x98badcfe;
  d = 0x10325476;

  ROUND_1
  ROUND_2
  ROUND_3
  ROUND_4
  
  if (cached_hash_to_break_1  == a   &&
      cached_hash_to_break_2  == b   &&
      cached_hash_to_break_3  == c   &&
      cached_hash_to_break_4  == d
    ) {
    // cuPrintf("\n**************************found*****************\n\n");
    global_word[0] = x_0;
    global_word[1] = x_1 & 0xffff;
  }


  // if (threadIdx.x == 0 && offset_1+threadIdx.y == 0 && offset_2+blockIdx.x == 0 && offset_3+blockIdx.y == 0 && offset_4+blockIdx.z == 0) {
  // // if (threadIdx.x == 0) {
  //   cuPrintf("%c,%c,%c,%c,%c,%c\n", constant_ascii_codes[threadIdx.x], constant_ascii_codes[offset_1+threadIdx.y],
  //     constant_ascii_codes[offset_2+blockIdx.x], constant_ascii_codes[offset_3+blockIdx.y],
  //     constant_ascii_codes[offset_4+blockIdx.z], constant_ascii_codes[offset_5+blockIdx.z]);
  //   cuPrintf("1: %d - 2: %d - 3: %d - 4: %d - 5: %d\n", offset_1, offset_2, offset_3, offset_4, offset_5);
  //   cuPrintf("tx: %d - ty: %d - bx: %d - by: %d - bz: %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, blockIdx.z);
  //   cuPrintf("a: %u\n", a);
  //   cuPrintf("b: %u\n", b);
  //   cuPrintf("c: %u\n", c);
  //   cuPrintf("d: %u\n", d);
  // }

}

int main (int argc, char *argv[]) {
  if (argc != 2) {
    printf("**invalid number of arguments**\n");
    return 1;
  }
  char original_word[10];
  strcpy(original_word, argv[1]);

  int rv = 0;
  rv = brute_force(original_word);

  printf("Broke Hash? %s\n", rv == 1 ? "YES" : "NO");

  return 0;
}



int brute_force (char* original_word) {
  hipDeviceProp_t prop;
  int whichDevice;
  HANDLE_ERROR(hipGetDevice( & whichDevice));
  HANDLE_ERROR(hipGetDeviceProperties( & prop, whichDevice));
  if (!prop.deviceOverlap) {
    printf("Device will not handle overlaps, so no "
    "speed up from streams\n");
    return 0;
  }

  char hash_str[32];
  char h_hash[16];
  

  create_md5_hash_str(original_word, hash_str);

  fprintf(stdout, "original_word: |%s|\n", original_word);
  fprintf(stdout, "hash to break: %s\n", hash_str);

  break_down_hash(h_hash, hash_str);

  int* X = (int *)h_hash;
  printf("X[0]: %u\n", X[0]);
  printf("X[1]: %u\n", X[1]);
  printf("X[2]: %u\n", X[2]);
  printf("X[3]: %u\n\n\n", X[3]);

  // After the digest is finished those operations are made
  // Since we are the ones creating the digest there is no need perform those operations
  X[0] -= 0x67452301;
  X[1] -= 0xefcdab89;
  X[2] -= 0x98badcfe;
  X[3] -= 0x10325476;

  printf("X[0]: %u\n", X[0]);
  printf("X[1]: %u\n", X[1]);
  printf("X[2]: %u\n", X[2]);
  printf("X[3]: %u\n", X[3]);  

  // initialize the stream
  hipStream_t stream;
  HANDLE_ERROR(hipStreamCreate( & stream));


  char host_ascii_codes[64];
  char* host_word;
  // allocate page-locked memory, used to stream
  HANDLE_ERROR(hipHostAlloc((void**)&host_word, sizeof(char) * WORD_SIZE, hipHostMallocDefault));

  // int* device_hash_to_break;
  int* device_word;
  HANDLE_ERROR(hipMalloc((void**)&device_word, sizeof(char) * WORD_SIZE));

  int ascci_counter = 0;
  for (int i = 48; i <= 57; i++) {
    host_ascii_codes[ascci_counter++] = i;
  }
  for (int i = 65; i <= 90; i++) {
    host_ascii_codes[ascci_counter++] = i;
  }
  for (int i = 97; i <= 122; i++) {
    host_ascii_codes[ascci_counter++] = i;
  }
  host_ascii_codes[62] = 63; // ?
  host_ascii_codes[63] = 64; // @

  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(constant_ascii_codes), host_ascii_codes, sizeof(char) * 64));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(constant_hash_to_break), X, sizeof(int) * 4));

  hipEvent_t start, stop;
  float elapsedTime;
  // start the timers
  HANDLE_ERROR(hipEventCreate( & start));
  HANDLE_ERROR(hipEventCreate( & stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  // cudaPrintfInit();
  dim3 dimBlock(64, 16, 1);
  dim3 dimGrid(16, 16, 16);

  for (int it_1 = 0; it_1 < 64; it_1 += 16) { // ThreadIdx.y
    for (int it_2 = 0; it_2 < 64; it_2 += 16) { // BlockIdx.x
      for (int it_3 = 0; it_3 < 64; it_3 += 16) { // BlockIdx.y
        for (int it_4 = 0; it_4 < 64; it_4 += 16) { // BlockIdx.z
          for (int it_5 = 0; it_5 < 64; it_5 += 1) { 
            kernel <<<dimGrid, dimBlock, 0, stream>>>(device_word,
              it_1, it_2,
              it_3, it_4,
              it_5);
            // copy the data from device to locked memory
            HANDLE_ERROR(hipMemcpyAsync(host_word, device_word, sizeof(char) * WORD_SIZE, hipMemcpyDeviceToHost, stream));
          }
          // printf("after loop\n");
          // copy result chunk from locked to full buffer
          HANDLE_ERROR(hipStreamSynchronize(stream));
          // printf("after sync.\n");

          HANDLE_ERROR(hipEventRecord(stop, 0));
          HANDLE_ERROR(hipEventSynchronize(stop));
          HANDLE_ERROR(hipEventElapsedTime( & elapsedTime, start, stop));
          printf("[%d, %d, %d, %d] Time taken:  %3.1f ms\n", it_1, it_2, it_3, it_4, elapsedTime);
          

          int broke = 1;
          for (int j = 0; j < WORD_SIZE; j++) {
            if (original_word[j] != host_word[j]) {
              // printf("Did not find a match, going to next iteration.\n");
              broke = 0;
              break;
            }
          }

          if (broke) {
            printf("Found a match!\n");
            for (int i = 0; i < WORD_SIZE; i++) {
              printf("%c,", host_word[i]);
            }
            printf("|\n");
            HANDLE_ERROR(hipEventRecord(stop, 0));
            HANDLE_ERROR(hipEventSynchronize(stop));
            HANDLE_ERROR(hipEventElapsedTime( & elapsedTime, start, stop));
            printf("[%d, %d] Time taken:  %3.1f ms\n", it_1, it_2, elapsedTime);
            return 1;
          }
        } // End Loop 4
      } // End Loop 3
    } // End Loop 2
  } // End Loop 1

  // cudaPrintfDisplay(stdout, true);
  // cudaPrintfEnd();


  // cleanup the streams and memory
  HANDLE_ERROR(hipHostFree(host_word));
  HANDLE_ERROR(hipFree(device_word));
  HANDLE_ERROR(hipStreamDestroy(stream));
  return 0;  
}




// Create hash string for |word|
// This is the hash string to the original hash we are trying to break
// We use the hash string to create the 16 8bit hexadecinal chunks
void create_md5_hash_str(const char* word, char* hash_str) {
  int di;

  md5_state_t state;
  md5_byte_t digest[16];  

  md5_init(&state);
  md5_append(&state, (const md5_byte_t *)word, strlen(word));
  md5_finish(&state, digest);
  
  for (di = 0; di < 16; di++) {
    sprintf(hash_str + di * 2, "%02x", digest[di]); 
  }
}

void break_down_hash (char* hash, char* hash_str) {
  int i, j;
  int digest = 0;
  int dec1 = 0;
  int dec2 = 0;
  for (i = 0, j = 0; i < 32; i += 2, j++) {
    dec1 = hex_to_decimal(hash_str[i]);
    dec2 = hex_to_decimal(hash_str[i+1]);
    // fprintf(stdout, "dec1: %d\n", dec1);
    // fprintf(stdout, "dec2: %d\n", dec2);
    digest =  dec1 * 16 + dec2;
    hash[j] = digest;
    // fprintf(stdout, "i %d - digest: %d\n", i, digest);
  }
}

int hex_to_decimal (char c) {
  switch (c) {
    case '0':
      return 0;
    case '1':
      return 1;
    case '2':
      return 2;
    case '3':
      return 3;
    case '4':
      return 4;
    case '5':
      return 5;
    case '6':
      return 6;
    case '7':
      return 7;
    case '8':
      return 8;
    case '9':
      return 9;
    case 'a':
    case 'A':
      return 10;
    case 'b':
    case 'B':
      return 11;
    case 'c':
    case 'C':
      return 12;
    case 'd':
    case 'D':
      return 13;
    case 'e':
    case 'E':
      return 14;
    case 'f':
    case 'F':
      return 15;
    default:
      fprintf(stdout, "FAILED to get convert %c to decimal\n", c);
  }

  return -1;
}







